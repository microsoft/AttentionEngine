// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<{{cutlass_dtype}}, {{dimqk}}, {{dimv}}>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_qkdim{{dimqk}}_vdim{{dimv}}<{{cutlass_dtype}}>(params, stream);
}
